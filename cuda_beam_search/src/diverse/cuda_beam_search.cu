#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void process_diverse_beam_search_kernel(
    const int64_t* input_ids,
    const float* next_scores,
    const int64_t* next_tokens,
    const int64_t* next_indices,
    float* next_beam_scores,
    int64_t* next_beam_tokens,
    int64_t* next_beam_indices,
    const int batch_size,
    const int num_beams,
    const int num_beam_groups,
    const int vocab_size,
    const int pad_token_id,
    const int eos_token_id,
    const float diversity_penalty
) {
    const int batch_idx = blockIdx.x;
    const int group_idx = blockIdx.y;
    const int beam_idx = threadIdx.x;
    
    const int group_size = num_beams / num_beam_groups;
    if (beam_idx >= group_size) return;
    
    // Get scores for this group
    float* group_scores = new float[vocab_size];
    for (int i = 0; i < vocab_size; i++) {
        group_scores[i] = next_scores[batch_idx * vocab_size + i];
    }
    
    // Apply diversity penalty
    if (group_idx > 0 && diversity_penalty > 0.0) {
        for (int prev_group = 0; prev_group < group_idx; prev_group++) {
            for (int prev_beam = 0; prev_beam < group_size; prev_beam++) {
                const int prev_idx = batch_idx * num_beams + prev_group * group_size + prev_beam;
                const int64_t prev_token = next_beam_tokens[prev_idx];
                
                // Find where this token appears in next_tokens
                for (int i = 0; i < vocab_size; i++) {
                    if (next_tokens[batch_idx * vocab_size + i] == prev_token) {
                        group_scores[i] -= diversity_penalty;
                    }
                }
            }
        }
    }
    
    // Find the best token for this beam
    float best_score = -1e9;
    int64_t best_token = pad_token_id;
    int64_t best_index = 0;
    
    for (int i = 0; i < vocab_size; i++) {
        if (group_scores[i] > best_score) {
            best_score = group_scores[i];
            best_token = next_tokens[batch_idx * vocab_size + i];
            best_index = next_indices[batch_idx * vocab_size + i];
        }
    }
    
    // Store the results
    const int out_idx = batch_idx * num_beams + group_idx * group_size + beam_idx;
    next_beam_scores[out_idx] = best_score;
    next_beam_tokens[out_idx] = best_token;
    next_beam_indices[out_idx] = best_index;
    
    delete[] group_scores;
}

std::vector<torch::Tensor> process_diverse_beam_search_cuda(
    torch::Tensor input_ids,
    torch::Tensor next_scores,
    torch::Tensor next_tokens,
    torch::Tensor next_indices,
    int num_beam_groups,
    float diversity_penalty,
    int pad_token_id,
    int eos_token_id
) {
    CHECK_INPUT(input_ids);
    CHECK_INPUT(next_scores);
    CHECK_INPUT(next_tokens);
    CHECK_INPUT(next_indices);
    
    const int batch_size = next_scores.size(0);
    const int num_beams = input_ids.size(0) / batch_size;
    const int vocab_size = next_scores.size(1);
    
    auto next_beam_scores = torch::zeros({batch_size, num_beams}, 
        torch::dtype(torch::kFloat32).device(input_ids.device()));
    auto next_beam_tokens = torch::zeros({batch_size, num_beams}, 
        torch::dtype(torch::kInt64).device(input_ids.device()));
    auto next_beam_indices = torch::zeros({batch_size, num_beams}, 
        torch::dtype(torch::kInt64).device(input_ids.device()));
    
    const dim3 blocks(batch_size, num_beam_groups);
    const dim3 threads(num_beams / num_beam_groups);
    
    process_diverse_beam_search_kernel<<<blocks, threads>>>(
        input_ids.data_ptr<int64_t>(),
        next_scores.data_ptr<float>(),
        next_tokens.data_ptr<int64_t>(),
        next_indices.data_ptr<int64_t>(),
        next_beam_scores.data_ptr<float>(),
        next_beam_tokens.data_ptr<int64_t>(),
        next_beam_indices.data_ptr<int64_t>(),
        batch_size,
        num_beams,
        num_beam_groups,
        vocab_size,
        pad_token_id,
        eos_token_id,
        diversity_penalty
    );
    
    return {next_beam_scores, next_beam_tokens, next_beam_indices};
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("process_diverse_beam_search", &process_diverse_beam_search_cuda, "Process diverse beam search (CUDA)");
} 